#include "hip/hip_runtime.h"
/*
 * Copyright 2020, NVIDIA CORPORATION.
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <stdint.h>
#include <stdio.h>

#include "nvbit_reg_rw.h"
#include "utils/utils.h"
#include "pf_injector.h"
#include "arch.h"


extern "C" __device__ __noinline__ void inject_error(uint64_t piinfo, uint64_t pverbose_device, int destGPRNum, int regval, int numDestGPRs, int maxRegs) {

				inj_info_t* inj_info = (inj_info_t*)piinfo; 
				uint32_t verbose_device = *((uint32_t *)pverbose_device);
				
				//check performed on the Straming Multiprocessor ID
				uint32_t smid;
				asm("mov.u32 %0, %smid;" :"=r"(smid));
				if (smid != inj_info->injSMID) 
							return; // This is not the selected SM. No need to proceed.
				
				//not used
				uint32_t nctaidX;
				asm("mov.u32 %0, %nctaid.x;" :"=r"(nctaidX));
				uint32_t nctaidY;
				asm("mov.u32 %0, %nctaid.y;" :"=r"(nctaidY));
				uint32_t nctaidZ;
				asm("mov.u32 %0, %nctaid.z;" :"=r"(nctaidZ));								
				uint32_t ctaidX;
				asm("mov.u32 %0, %ctaid.x;" :"=r"(ctaidX));
				uint32_t ctaidY;
				asm("mov.u32 %0, %ctaid.y;" :"=r"(ctaidY));			
				uint32_t ctaidZ;
				asm("mov.u32 %0, %ctaid.z;" :"=r"(ctaidZ));
				
				
				//Thread ID calculation
				uint32_t threadidX;
				asm("mov.u32 %0, %tid.x;" :"=r"(threadidX));
				uint32_t threadidY;
				asm("mov.u32 %0, %tid.y;" :"=r"(threadidY));
				uint32_t threadidZ;
				asm("mov.u32 %0, %tid.z;" :"=r"(threadidZ));
				uint32_t nthreadidX;
				asm("mov.u32 %0, %ntid.x;" :"=r"(nthreadidX));
				uint32_t nthreadidY;
				asm("mov.u32 %0, %ntid.y;" :"=r"(nthreadidY));
				uint32_t nthreadidZ;
				asm("mov.u32 %0, %ntid.z;" :"=r"(nthreadidZ));
				uint32_t threadid = (threadidZ*nthreadidY*nthreadidX)+(threadidY*nthreadidX)+threadidX;		
	
			
				//check if the thread ID is the same of fault
				if (threadid != inj_info->injThreadID) 					
					return;
				assert(numDestGPRs > 0);
				uint32_t injAfterVal = 0; 
				uint32_t injBeforeVal = nvbit_read_reg(destGPRNum); // read the register value
				if (destGPRNum != inj_info->injReg ) { //ADDED
								injAfterVal = injBeforeVal;
				
				} else {
					if(inj_info->injStuckat == 1){
								injAfterVal = injBeforeVal | (inj_info->injMask); //OR
								nvbit_write_reg(destGPRNum, injAfterVal);
					}
					else {	
								injAfterVal = injBeforeVal & (~inj_info->injMask);//Bug coredumped					
								//injAfterVal = ~injBeforeVal ^ (inj_info->injMask); //NOR				
								nvbit_write_reg(destGPRNum, injAfterVal);
											
					}

				}
				// updating counter/flag to check whether the error was injected
				if (verbose_device) printf("register=%d, before=0x%x, after=0x%x, expected_after=0x%x, mask=0x%x\n", destGPRNum, injBeforeVal, nvbit_read_reg(destGPRNum), injAfterVal, inj_info->injMask);
				inj_info->errorInjected = true; 
				atomicAdd((unsigned long long*) &inj_info->injNumActivations, 1LL);  
}

